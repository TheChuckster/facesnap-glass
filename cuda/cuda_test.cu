
#include <hip/hip_runtime.h>
#include <stdio.h>
 
const int N = 16;
 
__global__ 
void hello(char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}
 
int cuda_test()
{
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 
	char* ad = NULL;
	int* bd = NULL;
 
	printf("%s", a);
 
	hipMalloc((void**)&ad, sizeof(a)); 
	hipMalloc((void**)&bd, sizeof(b)); 

	hipMemcpy(ad, a, sizeof(a), hipMemcpyHostToDevice); 
	hipMemcpy(bd, b, sizeof(b), hipMemcpyHostToDevice); 
	
	dim3 dimBlock(N, 1);
	dim3 dimGrid(1, 1);
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy(a, ad, sizeof(a), hipMemcpyDeviceToHost); 

	hipFree(ad);
	hipFree(bd);
	
	printf("%s\n", a);
	
	return EXIT_SUCCESS;
}
